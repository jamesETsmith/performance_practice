
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8

void cpuPrinter(int nlim) {
  for (int idx = 0; idx < nlim; idx++) printf("CPU Prints Idx: %d\n", idx);

  printf("\n");
}

__global__ void gpuPrinter(void) {
  int idx =
      threadIdx.x +
      blockIdx.x * blockDim.x; /* The local thread index (threadIdx.x) in the
                                  right hand side should be shifted by an offset
                                  value (blockIdx.x * blockDim.x) to compensate
                                  translate it to a global index */
  printf("GPU Prints Idx: %d\n",
         idx); /* Write the kernel for individual threads */
}

int main(int argc, char **argv) {
  cpuPrinter(N);

  gpuPrinter<<<2, N / 2>>>(); /*  Organizing eight threads in two blocks with
                                 four threads */

  hipDeviceSynchronize();

  return (EXIT_SUCCESS);
}