
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8

void cpuPrinter(int nlim) {
  for (int idx = 0; idx < nlim; idx++) printf("CPU Prints Idx: %d\n", idx);

  printf("\n");
}

__global__ void gpuPrinter(void) {
  int idx = threadIdx.x;
  printf("GPU Prints Idx: %d\n",
         idx); /* Write the kernel for individual threads */
}

int main(int argc, char **argv) {
  cpuPrinter(N);

  gpuPrinter<<<1, N>>>(); /*  Launch the kernel for many threads */
                          /*  CUDA will raise an error if N > 1024 */
  hipDeviceSynchronize();

  return (EXIT_SUCCESS);
}